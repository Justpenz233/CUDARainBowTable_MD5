#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdlib>
#include <cstdio>
#include <stdint.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <cstring>
#include <ctime>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>

#define MAXTHREAD MaxSM*MaxSP

#define MAXCreate 100
//How many RainBow Table do you want to create
#define BlockNum 4194304
//The max num in per block

//md5:a 128 bit,standard
//md5str:32 chars--->when input,we pack it into the standard version
//when we calculate or work or save or we input from the table file we saved we use the standard version
//when we ouput or input from the stdin the md5,we use the md5str

#define MinC  '0'
#define MaxC  '9'
#define CHCNT 10
#define SPCH 0
#define CHAINLEN 1200
#define RDCTCNT 4

using namespace std;

int DeviceNum;
int MaxSP, MaxSM;
char Begin[BlockNum * 8 + 1], End[BlockNum * 8 + 1];
int begin_len[BlockNum + 1], end_len[BlockNum + 1];

__device__ void MD5(char *src, int src_len, uint32_t *rslt) { //size_t
//the md5 will store at rslt
// leftrotate function definition
//attention:the strlen(src) should <=8
#define LEFTROTATE(x, c) (((x) << (c)) | ((x) >> (32 - (c))))

	// Message (to prepare)
	uint8_t tmp[128];//lilun shang zhiyao 64,but wanyi...
	uint8_t *msg = tmp;
	uint16_t prei;

	// Note: All variables are unsigned 32 bit and wrap modulo 2^32 when calculating

	// r specifies the per-round shift amounts

	uint32_t r[] = {7, 12, 17, 22, 7, 12, 17, 22, 7, 12, 17, 22, 7, 12, 17, 22,
	                5,  9, 14, 20, 5,  9, 14, 20, 5,  9, 14, 20, 5,  9, 14, 20,
	                4, 11, 16, 23, 4, 11, 16, 23, 4, 11, 16, 23, 4, 11, 16, 23,
	                6, 10, 15, 21, 6, 10, 15, 21, 6, 10, 15, 21, 6, 10, 15, 21
	               };

	// Use binary integer part of the sines of integers (in radians) as constants// Initialize variables:
	uint32_t k[] = {
		0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee,
		0xf57c0faf, 0x4787c62a, 0xa8304613, 0xfd469501,
		0x698098d8, 0x8b44f7af, 0xffff5bb1, 0x895cd7be,
		0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821,
		0xf61e2562, 0xc040b340, 0x265e5a51, 0xe9b6c7aa,
		0xd62f105d, 0x02441453, 0xd8a1e681, 0xe7d3fbc8,
		0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed,
		0xa9e3e905, 0xfcefa3f8, 0x676f02d9, 0x8d2a4c8a,
		0xfffa3942, 0x8771f681, 0x6d9d6122, 0xfde5380c,
		0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70,
		0x289b7ec6, 0xeaa127fa, 0xd4ef3085, 0x04881d05,
		0xd9d4d039, 0xe6db99e5, 0x1fa27cf8, 0xc4ac5665,
		0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039,
		0x655b59c3, 0x8f0ccc92, 0xffeff47d, 0x85845dd1,
		0x6fa87e4f, 0xfe2ce6e0, 0xa3014314, 0x4e0811a1,
		0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391
	};

	(*(rslt + 0)) = 0x67452301;
	(*(rslt + 1)) = 0xefcdab89;
	(*(rslt + 2)) = 0x98badcfe;
	(*(rslt + 3)) = 0x10325476; //the magic number

	int new_len = 56; //as the max_length is 8,the new_len must be 56
	//place soruce data
	//add a bit of 1
	//add bits of 0 until have 448 bits
	//the rest
	for (prei = 0; prei < src_len; prei++) *(msg + prei) = *(src + prei);
	*(msg + prei++) = 128; // write the "1" bit

	while (prei < new_len)
		*(msg + prei++) = 0;
	*(msg + prei++) = 8 * src_len;
	while (prei < 128) //lilun shang zhixuyao 64,but wanyi...
		*(msg + prei++) = 0;

	// Process the message in successive 512-bit chunks:
	//for each 512-bit chunk of message:
	int offset;
	for (offset = 0; offset < new_len; offset += (512 / 8)) {

		// break chunk into sixteen 32-bit words w[j], 0 ≤ j ≤ 15
		uint32_t *w = (uint32_t *) (msg + offset);

#ifdef DEBUG
		printf("offset: %d %x\n", offset, offset);

		int j;
		for (j = 0; j < 64; j++) printf("%x ", ((uint8_t *) w)[j]);
		puts("");
#endif

		// Initialize hash value for this chunk:
		uint32_t a = (*(rslt + 0));
		uint32_t b = (*(rslt + 1));
		uint32_t c = (*(rslt + 2));
		uint32_t d = (*(rslt + 3));

		// Main loop:
		uint32_t i;
		for (i = 0; i < 64; i++) {

#ifdef ROUNDS
			uint8_t *p;
			printf("%i: ", i);
			p = (uint8_t *)&a;
			printf("%2.2x%2.2x%2.2x%2.2x ", p[0], p[1], p[2], p[3], a);

			p = (uint8_t *)&b;
			printf("%2.2x%2.2x%2.2x%2.2x ", p[0], p[1], p[2], p[3], b);

			p = (uint8_t *)&c;
			printf("%2.2x%2.2x%2.2x%2.2x ", p[0], p[1], p[2], p[3], c);

			p = (uint8_t *)&d;
			printf("%2.2x%2.2x%2.2x%2.2x", p[0], p[1], p[2], p[3], d);
			puts("");
#endif


			uint32_t f, g;

			if (i < 16) {
				f = (b & c) | ((~b) & d);
				g = i;
			} else if (i < 32) {
				f = (d & b) | ((~d) & c);
				g = (5 * i + 1) % 16;
			} else if (i < 48) {
				f = b ^ c ^ d;
				g = (3 * i + 5) % 16;
			} else {
				f = c ^ (b | (~d));
				g = (7 * i) % 16;
			}

#ifdef ROUNDS
			printf("f=%x g=%d w[g]=%x\n", f, g, w[g]);
#endif
			uint32_t temp = d;
			d = c;
			c = b;
			b = b + LEFTROTATE((a + f + k[i] + w[g]), r[i]);
			a = temp;
		}

		// Add this chunk's hash to result so far:

		(*(rslt + 0)) += a;
		(*(rslt + 1)) += b;
		(*(rslt + 2)) += c;
		(*(rslt + 3)) += d;
	}
}
__device__ void work(char end) {

}

__device__ void rdct(int step_id, const char *md5, char *pwd, int *len)
{	//md5 reduct to a passwd,save to pwd. and the length save to len
	//int id;//??? need li si qi jie jue
	unsigned long long hash = 0ll;
	unsigned int i;
	unsigned int *h1;
	switch (step_id % RDCTCNT) {
	case 0:
		for (i = 0; i < 16; i++)
			hash = hash * 131 + (*(md5 + i));
		break;
	case 1:
		h1 = (unsigned int*)(&hash);
		i = *((unsigned int*)(md5));
		i = ~i + (i << 15);
		i = i ^ (i >> 12);
		i = i + (i << 2);
		i = i ^ (i >> 4);
		i = i * 2057;
		i = i ^ (i >> 16);
		*h1 = i; //get the high 32bits
		i = *((unsigned int*)(md5 + 8));
		i = ~i + (i << 15);
		i = i ^ (i >> 12);
		i = i + (i << 2);
		i = i ^ (i >> 4);
		i = i * 2057;
		i = i ^ (i >> 16);
		*(h1 + 1) = i;
		break;
	case 2:
		h1 = (unsigned int*)(&hash);
		i = *((unsigned int*)(md5 + 4));
		i = ~i + (i << 15);
		i = i ^ (i >> 12);
		i = i + (i << 2);
		i = i ^ (i >> 4);
		i = i * 2057;
		i = i ^ (i >> 16);
		*h1 = i; //get the high 32bits
		i = *((unsigned int*)(md5 + 12));
		i = ~i + (i << 15);
		i = i ^ (i >> 12);
		i = i + (i << 2);
		i = i ^ (i >> 4);
		i = i * 2057;
		i = i ^ (i >> 16);
		*(h1 + 1) = i;
		break;
	case 3:
		uint16_t *p = (uint16_t *)md5;
		*len = 8;
		for (int i = 0; i < 8; i++) *(pwd + i) = (char)(((*(p + i)**(p + i) * 163) % 19163 % CHCNT) + MinC);
		return;
	}
	for (i = 0; hash && i < 8; i++) {
		*(pwd + i) = (char)((hash % CHCNT) + MinC);
		hash /= CHCNT;
	}
	*len = i;
	for (i = 0; i < 8; i++) *(pwd + i) = SPCH;
}

__global__ void CreateRainBow_chain(char *ed, int *len) {
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	//now i assum id is the id th passwd
	char md5[16];//i don konw whether it
	for (int i = 0; i < CHAINLEN; i++) {
		MD5(ed + id * 8, *(len + id), (uint32_t*)md5);
		rdct(i, md5, ed, len);
	}
}
/*__device__ void printToFile() {

}*/

__host__ inline void print_to_file(int fi)
{
	char name[5];
	sprintf_s(name, "%.04d", fi);
	ofstream out(name, ios::binary);
	out.write(End, BlockNum * sizeof(char) * 8);
	out.close();
}

__host__ inline void randBegin() {

	for (int id = 0; id < BlockNum; id++) {
		int len = rand() % 8 + 1;
		if (len <= 3) len = rand() % 8 + 1; //add by lly
		end_len[id] = begin_len[id] = (uint8_t)len;
		int i;
		for (i = 0; i < len; i++) {
			End[id * 8 + i] = Begin[id * 8 + i] = (char)(rand() % (MaxC - MinC) + MinC);
		}
		for (; i < 8; i++) End[id * 8 + i] = Begin[id * 8 + i] = 0;
	}

}

int main()
{
	int num;
	hipDeviceProp_t prop;
	hipGetDeviceCount(&num);
	for (int i = 0; i < num; i++)
	{
		hipGetDeviceProperties(&prop, i);
		MaxSM = prop.multiProcessorCount;
		MaxSP = prop.maxThreadsPerBlock;
	}
	dim3 grid(MaxSM, 1, 1), block(MaxSP, 1, 1);

	srand((unsigned)time(NULL));
	randBegin();//get random begin,save at the array:Begin

	for (int i = 0; i <= MAXCreate / MAXTHREAD; i++) {
		char *tring_dev; int *temp; int *len;
		hipMalloc((void **)&temp, sizeof(int));
		hipMalloc((void **)&tring_dev, BlockNum * 8 * sizeof(char));
		hipMalloc((void **)&len, BlockNum * sizeof(uint8_t));

		hipMemcpy(temp, &i, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(tring_dev, End, BlockNum * 8 * sizeof(char), hipMemcpyHostToDevice);
		hipMemcpy(len, end_len, BlockNum * sizeof(uint8_t), hipMemcpyHostToDevice);

		CreateRainBow_chain <<< grid, block>>> (tring_dev, len);

		hipMemcpy(End, tring_dev, BlockNum * 8 * sizeof(char), hipMemcpyDeviceToHost);
		hipMemcpy(end_len, len, BlockNum * sizeof(uint8_t), hipMemcpyDeviceToHost);

		hipFree(tring_dev); hipFree(temp); hipFree(len);
		print_to_file(i);
		//printToFile();
	}

	system("pause");
}
